#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "cuda_check_error.h"

#include "SFML/Graphics.hpp"
#include "SFML/Graphics/Image.hpp"

#include <ctime>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#define _USE_MATH_DEFINES
#include <math.h>

#include <thread>
#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>


/*
The Complex Function We Are Visualizing:
f(z)=1.5^z^1.5j
*/

struct ComplexFunctor {
    __device__ hipDoubleComplex operator()(hipDoubleComplex z, double c1, double c2) const {

        if (z.x != 0.0 || z.y != 0.0) {
            hipDoubleComplex base = make_hipDoubleComplex(c1, 0.0);
            hipDoubleComplex exponent = make_hipDoubleComplex(0.0, c2);

            // Calculate z^(2.0j)
            double r = hipCabs(z);
            double theta = atan2(hipCimag(z), hipCreal(z));
            hipDoubleComplex log_z = make_hipDoubleComplex(log(r), theta);
            hipDoubleComplex temp = hipCmul(log_z, exponent);
            hipDoubleComplex z_pow_exponent = make_hipDoubleComplex(exp(temp.x) * cos(temp.y), exp(temp.x) * sin(temp.y));

            // Calculate 1.5^z_pow_exponent
            r = hipCabs(base);
            theta = atan2(hipCimag(base), hipCreal(base));
            log_z = make_hipDoubleComplex(log(r), theta);
            temp = hipCmul(log_z, z_pow_exponent);
            hipDoubleComplex result = make_hipDoubleComplex(exp(temp.x) * cos(temp.y), exp(temp.x) * sin(temp.y));

            return result;
        } else {
            return make_hipDoubleComplex(0.0, 0.0);
        }
        
    }
};

/*
Populating Pixels with Function Outputs
*/

__device__ void populatePixelWithFunctionOutput(hipDoubleComplex *value, double *phase, double *mag, int m, int n, double x, double y, int i, int j, int scale, ComplexFunctor functor, double c1, double c2) {
    double u = x + (i - m / 2.0) * (1.0 / scale);
    double v = y + (j - n / 2.0) * (1.0 / scale);
    hipDoubleComplex z = make_hipDoubleComplex(u, v);
    hipDoubleComplex va = functor(z,c1,c2);

    double ph = atan2(hipCimag(va), hipCreal(va));
    double ma = hipCabs(va);

    value[i * n + j] = va;
    phase[i * n + j] = ph;
    mag[i * n + j] = ma;
}

__global__ void populatePixelWithFunctionOutputKernel(hipDoubleComplex *value, double *phase, double *mag, int m, int n, double x, double y, int scale, ComplexFunctor functor,double c1, double c2) {
    int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y;

    int di = m / (gridDim.x*blockDim.x);
    int dj = n / (gridDim.y*blockDim.y);

    for (int i = i0; i < i0+di; i++){
        for (int j = j0; j < j0+dj; j++){
            populatePixelWithFunctionOutput(value,phase,mag,m,n,x,y,i,j,scale,functor,c1,c2);
        }
    }
}

/*
Populating Pixels with Numerical Derivative of Function
*/
__device__ void populatePixelWithNumericalDerivativeOutput(hipDoubleComplex *value, hipDoubleComplex *dvalue, double *dphase, double* dmag, int m, int n, int i, int j, int scale) 
{

    int idx = i * n + j;

    hipDoubleComplex prev = (i-1 < 0 || j-1 < 0) ? value[idx] : value[(i-1) * n + (j-1)];
    hipDoubleComplex next = (i+1 >= m || j+1 >= n) ? value[idx] : value[(i+1) * n + (j+1)];

    float a = 2.0f * (1.0f / scale);
    float b = 2.0f * (1.0f / scale);

    // Calculate dv
    hipDoubleComplex dv = make_hipDoubleComplex((next.x - prev.x) / std::sqrt(a * a + b * b), 
                                              (next.y - prev.y) / std::sqrt(a * a + b * b));
    
    // Calculate dp using atan2
    double dp = std::atan2(hipCimag(dv), hipCreal(dv));
    
    // Calculate dm as the magnitude of dv
    double dm = hipCabs(dv);

    dvalue[i*n+j]=dv;
    dphase[i*n+j]=dp;
    dmag[i*n+j]=dm;
}

__global__ void populatePixelWithNumericalDerivativeOutputKernel(hipDoubleComplex *value, hipDoubleComplex *dvalue, double *dphase, double *dmag, int m, int n, int scale) {
    int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y;

    int di = m / (gridDim.x*blockDim.x);
    int dj = n / (gridDim.y*blockDim.y);

    for (int i = i0; i < i0+di; i++){
        for (int j = j0; j < j0+dj; j++){
            populatePixelWithNumericalDerivativeOutput(value,dvalue,dphase,dmag,m,n,i,j,scale);
        }
    }
}

/*
Create a finalized HeightMap
def colorbinHeightcontinuousPixel(colorbinHeightcontinuous: np.ndarray, 
                                  phase:np.ndarray, phasemap:np.ndarray, Np:int, maxphase: float, minphase: float, 
                                  dphase:np.ndarray, maxdphase: float, mindphase: float, 
                                  m:int, n:int, i:int, j:int):
    magbinPixel(colorbinHeightcontinuous,phase,m,n,i,j,phasemap,Np,maxphase,minphase)
    colorbinHeightcontinuous[i*n + j]+=((dphase[i*n+j] - mindphase)/(maxdphase-mindphase))

*/

__device__ double magbinPixel(double* mag, double* phase, int m, int n, int i, int j, double* magmap, int Nm, double maxmag, double minmag){
    int idx=(int)(((mag[i*n+j] - minmag)/(maxmag-minmag))*Nm);
    idx=min(Nm-1,idx);
    idx=max(0,idx);
    mag[i*n+j]=magmap[idx];
    mag[i*n+j]=0.0;
}

__device__ void colorbinHeightcontinuousPixel(double* colorbinHeightcontinuous, 
                                                    double* phase, double* phasemap, int Np, double maxphase, double minphase,
                                                    double* dphase, double maxdphase, double mindphase,
                                                    int m, int n, int i, int j) {
    magbinPixel(colorbinHeightcontinuous,phase,m,n,i,j,phasemap,Np,maxphase,minphase);
    colorbinHeightcontinuous[i*n + j]+=((dphase[i*n+j] - mindphase)/(maxdphase-mindphase));
}

__global__ void colorbinHeightcontinuousPixelKernel(double* colorbinHeightcontinuous, 
                                                    double* phase, double* phasemap, int Np, double maxphase, double minphase,
                                                    double* dphase, double maxdphase, double mindphase,
                                                    int m, int n) {
    int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y;

    int di = m / (gridDim.x*blockDim.x);
    int dj = n / (gridDim.y*blockDim.y);

    for (int i = i0; i < i0+di; i++){
        for (int j = j0; j < j0+dj; j++){
            colorbinHeightcontinuousPixel(colorbinHeightcontinuous,phase, phasemap, Np, maxphase, minphase, dphase, maxdphase, mindphase, m, n, i, j);
        }
    }
}

/*
Update GL Texture with image array
*/

// Define an artistic colormap with 20 colors grouped by similarity
__device__ uchar3 artisticColormap[10] = {
   // Blues/Purples/Aquas
        {236.0, 244.0, 214.0},   // Soft Green
        {154.0, 208.0, 194.0},   // Aquamarine
        {45.0, 149.0, 150.0},    // Teal
        {38.0, 80.0, 115.0},     // Deep Sky Blue
        {34.0, 9.0, 44.0},       // Dark Purple
        {135.0, 35.0, 65.0},     // Crimson
        {190.0, 49.0, 68.0},     // Raspberry
        {240.0, 89.0, 65.0},     // Coral
        {7.0, 102.0, 173.0},     // Cobalt Blue
        {41.0, 173.0, 178.0}     // Turquoise
};


// Function to map hue to RGB using artistic colormap
__device__ uchar3 hueToRGB(float hue, double maxphase, double minphase, int Nb) {
    // Calculate the bin index
    int bin = static_cast<int>((hue - minphase) / (maxphase - minphase) * (Nb - 1));

    // Ensure bin index is within bounds
    if (bin < 0) bin = 0;
    if (bin >= Nb) bin = Nb - 1;

    return artisticColormap[bin];
}

__global__ void update_surface_alt(hipSurfaceObject_t surface, double *value, double* phase, int m, int n, double maxphase, double minphase, int Nb)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    float hue = 255*static_cast<float>(phase[i * n + j]);
    float brighten = 255*static_cast<float>(value[i * n + j]);
    uchar3 rgb = hueToRGB(hue, maxphase, minphase, Nb);

    // Adjust brightness
    int r = static_cast<int>(rgb.x+brighten - 170);
    int g = static_cast<int>(rgb.y+brighten - 170);
    int b = static_cast<int>(rgb.z+brighten - 170);

    // Clamp the values between 0 and 255
    r = min(max(r, 0), 255);
    g = min(max(g, 0), 255);
    b = min(max(b, 0), 255);

    uchar4 pixel = make_uchar4(r, g, b, 0xff);
    surf2Dwrite(pixel, surface, j * sizeof(uchar4), i);
  
}

int main(int argc, char **argv)
{
    int m = 1440;
    int n = 2560;

    //int m = 512;
    //int n = 1024;
    
    sf::RenderWindow window(sf::VideoMode(n,m), "cuda_gl_interop");

    window.setFramerateLimit(60);
    window.setVerticalSyncEnabled(true);


    sf::Sprite sprite;
    sf::Texture txture;
    txture.create(n,m);
    
    hipArray *bitmap_d;

    GLuint gl_tex_handle = txture.getNativeHandle();

    hipGraphicsResource *cuda_tex_handle;

    hipGraphicsGLRegisterImage(&cuda_tex_handle, gl_tex_handle, GL_TEXTURE_2D,
                                hipGraphicsRegisterFlagsNone);
    cudaCheckError();

    hipGraphicsMapResources(1, &cuda_tex_handle, 0);
    cudaCheckError();

    hipGraphicsSubResourceGetMappedArray(&bitmap_d, cuda_tex_handle, 0, 0);
    cudaCheckError();

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    resDesc.res.array.array = bitmap_d;
    hipSurfaceObject_t bitmap_surface = 0;
    hipCreateSurfaceObject(&bitmap_surface, &resDesc);
    cudaCheckError();

    sprite.setTexture(txture);

    // Allocate computation memory
    hipDoubleComplex* value;
    double* phase;
    double* mag;
    hipMalloc(&value,m*n*sizeof(hipDoubleComplex));
    hipMalloc(&phase,m*n*sizeof(double));
    hipMalloc(&mag,m*n*sizeof(double));

    hipDoubleComplex* dvalue;
    double* dphase;
    double* dmag;
    hipMalloc(&dvalue,m*n*sizeof(hipDoubleComplex));
    hipMalloc(&dphase,m*n*sizeof(double));
    hipMalloc(&dmag,m*n*sizeof(double));

    double* colorbinHeightcontinuous;
    hipMalloc(&colorbinHeightcontinuous,m*n*sizeof(double));

    double x = 0.0;
    double y = 0.0;
    int scale = 100;

    int diff=5;

    double c1 = 1.5;
    double c2 = 1.5;

    // Main loop
    while (window.isOpen()) {
        // Event processing
        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
            }
        }

        // Check for escape key to exit the loop
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
            window.close();
            break;
        }

        // Move x and y based on key press
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::W)) {
            x -= 5/(double)scale;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::A)) {
            y -= 5/(double)scale;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::S)) {
            x += 5/(double)scale;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::D)) {
            y += 5/(double)scale;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::LShift)) {
            scale*=.99;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::RShift)) {
            scale*=1.01;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::Q)) {
            c1+=0.0009;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::E)) {
            c2+=0.0009;
        }
         if (sf::Keyboard::isKeyPressed(sf::Keyboard::R)) {
            c1-=0.0009;
        }
        if (sf::Keyboard::isKeyPressed(sf::Keyboard::T)) {
            c2-=0.0009;
        }
        
        dim3 pixelBlocks (m/32,n/32);
        dim3 pixelBlockSize (32,32);
        ComplexFunctor function;


        populatePixelWithFunctionOutputKernel<<<pixelBlocks,pixelBlockSize>>>(value,phase,mag,m,n,x,y,scale,function, c1, c2);
        hipDeviceSynchronize();

        populatePixelWithNumericalDerivativeOutputKernel<<<pixelBlocks,pixelBlockSize>>>(value,dvalue,dphase,dmag,m,n,scale);
        hipDeviceSynchronize();

        thrust::device_ptr<double> phase_ptr(phase);
        thrust::device_vector<double> phase_vector(phase_ptr, phase_ptr + m*n);
        double minphase = *thrust::min_element(phase_vector.begin(), phase_vector.end());
        double maxphase = *thrust::max_element(phase_vector.begin(), phase_vector.end());

        thrust::device_ptr<double> dphase_ptr(dphase);
        thrust::device_vector<double> dphase_vector(dphase_ptr, dphase_ptr + m*n);
        double mindphase = *thrust::min_element(dphase_vector.begin(), dphase_vector.end());
        double maxdphase = *thrust::max_element(dphase_vector.begin(), dphase_vector.end());

        // Output the results
        std::cout << "Minimum phase: " << minphase << std::endl;
        std::cout << "Maximum phase: " << maxphase << std::endl;
        std::cout << "Minimum dphase: " << mindphase << std::endl;
        std::cout << "Maximum dphase: " << maxdphase << std::endl;

        int Np = 10;
        std::vector<double> phasemap(Np);
        for (int i = 0; i < Np; ++i) {
            phasemap[i] = 0.8 * static_cast<double>(i) / Np;
        }
       
        double* phasemapdev;
        hipMalloc(&phasemapdev,Np*sizeof(double));
        hipMemcpy(phasemapdev,phasemap.data(),Np*sizeof(double),hipMemcpyHostToDevice);
        colorbinHeightcontinuousPixelKernel<<<pixelBlocks,pixelBlockSize>>>(colorbinHeightcontinuous,phase,phasemapdev,Np,maxphase,minphase,dphase,maxdphase,mindphase,m,n);
        hipDeviceSynchronize();

        //update_surface<<<pixelBlocks, pixelBlockSize>>>(bitmap_surface,value,m,n);
        update_surface_alt<<<pixelBlocks, pixelBlockSize>>>(bitmap_surface,colorbinHeightcontinuous,phase,m,n,minphase,maxphase,Np);
        hipDeviceSynchronize();

        //cudaCheckError();

        
        //cudaCheckError();

    
        window.clear();
        window.draw(sprite);
        window.display();

        printf("frame");
        //std::this_thread::sleep_for(std::chrono::milliseconds(100));
        //diff=(diff+5)%5;
    }

    return 0;
}